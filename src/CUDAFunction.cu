#include "hip/hip_runtime.h"
//*************************************************************************
//Copyright (c) 2020 IHEP                                                  
//Copyright (c) 2021 DESY                                                  
//This program is free software; you can redistribute it and/or modify     
//it under the terms of the GNU General Public License                     
//Author: chao li, li.chao@desy.de                                         
//*************************************************************************
#pragma once                                                             

#include <vector>
#include <complex>
#include <iostream>
#include<fstream>	
#include <numeric>
#include <cmath>
#include "Global.h"
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfft.h>
#include "CUDAFunction.cuh"

using namespace std;
using std::vector;
using std::complex;

__global__ void increment(double *partCord, double *oneTurnMap,  const int dim)
{
    // GPU only deal with matrix multiplication, and matrix is fixed.
}

void GPU_PartiOneTurnTransfer(double *partCord, double *oneTurnMap, const int dim)
{
    
    double *d_partCord,*d_oneTurnMap;
    hipMalloc((void**) &d_partCord, dim * sizeof(double) );
    hipMemcpy(d_partCord,partCord,dim * sizeof(double), hipMemcpyHostToDevice);
    
    hipMalloc((void**) &d_oneTurnMap, 36 * sizeof(double) );
    hipMemcpy(d_oneTurnMap,oneTurnMap,36 * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(1024);
    dim3 grid((dim + block.x -1)/ block.x);

    increment<<<grid, block>>>(d_partCord,d_oneTurnMap,dim);
    
    hipFree(d_partCord);
    hipFree(d_oneTurnMap);

    cout<<"test"<<endl;
    getchar();
}



