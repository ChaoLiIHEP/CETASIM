#include "hip/hip_runtime.h"
//*************************************************************************
//Copyright (c) 2020 IHEP                                                  
//Copyright (c) 2021 DESY                                                  
//This program is free software; you can redistribute it and/or modify     
//it under the terms of the GNU General Public License                     
//Author: chao li, li.chao@desy.de                                         
//*************************************************************************
#pragma once                                                             

#include <vector>
#include <complex>
#include <iostream>
#include<fstream>	
#include <numeric>
#include <cmath>
#include "Global.h"
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfft.h>
#include "Global.h"
#include "CUDAFunction.cuh"
#include "hip/hip_math_constants.h"


using namespace std;
using std::vector;
using std::complex;

__global__ void GPU_OneTurnMap(int partNum, double *partCord, double *ringPara )
{
    // partCord {x,px,y,py,z,pz,...}

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // from ringPara input
    double alpha[2]  = {ringPara[0],ringPara[1]};
    double beta[2]   = {ringPara[2],ringPara[3]};
    double nu[2]     = {ringPara[4],ringPara[5]};
    double chrom[2]  = {ringPara[6],ringPara[7]};
    double eta[2]    = {ringPara[8],ringPara[9]};
    double etap[2]   = {ringPara[10],ringPara[11]};
    double aDTX[2]   = {ringPara[12],ringPara[13]};
    double aDTY[2]   = {ringPara[14],ringPara[15]};
    double aDTXY[2]  = {ringPara[16],ringPara[17]};
    double alphac[3] = {ringPara[18],ringPara[19],ringPara[20]};
    double circRing  = ringPara[21];
  

    // // temp during calculation
    double amp[2],nutmp[2],phi[2],gamma[2],sinPhi[2],cosPhi[2];  // used in x and y direction
    double oneTurnMap[6][6];
    memset(oneTurnMap,0,sizeof(oneTurnMap));
    double vectX[6]={0,0,0,0,0,0};
    double vectY[6]={0,0,0,0,0,0};
    int partIndex = idx;


    if(partIndex < partNum)
    {
        #pragma unroll 
        for(int j=0;j<6;j++) vectX[j]  = partCord[partIndex * 6 + j];   // set the vectX (x,px,y,py,z,pz)
                     
        vectX[0] -= eta[0]  * vectX[5];   //x 
        vectX[1] -= etap[0] * vectX[5];   //px
        vectX[2] -= eta[1]  * vectX[5];   //y
        vectX[3] -= etap[1] * vectX[5];   //py

    
        #pragma unroll 
        for(int j=0;j<2;j++)
        {           
            amp[j]  = ( vectX[2*j] * vectX[2*j] + (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) * (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) ) / beta[j];
        }
        
        nutmp[0] = nu[0] + chrom[0] * vectX[5] +  aDTX[0] * amp[0] + aDTX[1] * amp[0] * amp[0]  / 2 + aDTXY[0] * amp[0] * amp[1];
        nutmp[1] = nu[1] + chrom[1] * vectX[5] +  aDTY[0] * amp[1] + aDTY[1] * amp[1] * amp[1]  / 2 + aDTXY[1] * amp[0] * amp[1];
        
        #pragma unroll
        for(int j=0;j<2;j++)
        {
            phi[j]      = 2.0  * M_PI * nutmp[j];
            sinPhi[j]   = sin( phi[j] );
            cosPhi[j]   = cos( phi[j] );
            gamma[j]    = (1 + alpha[j] * alpha[j] ) / beta[j];    
        }
        

        oneTurnMap[0][0] = cosPhi[0] + alpha[0] * sinPhi[0];
        oneTurnMap[0][1] =              beta[0] * sinPhi[0];
        oneTurnMap[1][0] =           - gamma[0] * sinPhi[0];
        oneTurnMap[1][1] = cosPhi[0] - alpha[0] * sinPhi[0];


        oneTurnMap[2][2] = cosPhi[1] + alpha[1] * sinPhi[1];
        oneTurnMap[2][3] =              beta[1] * sinPhi[1];
        oneTurnMap[3][2] =           - gamma[1] * sinPhi[1];
        oneTurnMap[3][3] = cosPhi[1] - alpha[1] * sinPhi[1];

        
        oneTurnMap[0][5] =  eta[0]  -  eta[0] * cosPhi[0] - (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[2][5] =  eta[1]  -  eta[1] * cosPhi[1] - (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];
        oneTurnMap[1][5] = -etap[0] - etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[3][5] = -etap[1] - etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][0] = -etap[0] + etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[4][2] = -etap[1] + etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][1] = eta[0]   - eta[0] * cosPhi[0] + (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[4][3] = eta[1]   - eta[1] * cosPhi[1] + (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];


        oneTurnMap[4][4] = 1;
        oneTurnMap[5][5] = 1;

        #pragma unroll
        for(int j=0;j<6;j++)
        {
            #pragma unroll
            for(int k=0;k<6;k++)
            {
                vectY[j] += oneTurnMap[j][k] * vectX[k];
            }
        }
        vectY[4] -= circRing * (alphac[0] * vectY[5]  + alphac[1] * vectY[5] * vectY[5] + alphac[2] * vectY[5] * vectY[5] * vectY[5] );
        
        #pragma unroll
        for(int j=0;j<6;j++) partCord[partIndex * 6 + j] = vectY[j];
    }
    
}

void GPU_PartiOneTurnTransfer(int macroEleNumPerBunch, double *partCord, int paraNum,double *ringPara)
{
    
    int partCordMem = macroEleNumPerBunch * 6 * sizeof(double);
    int paraMem     = paraNum *  sizeof(double); 
    double *d_partCord,*d_ringPara;
    
    hipMalloc((void**) &d_partCord, partCordMem);
    hipMalloc((void**) &d_ringPara, paraMem);
    hipMemcpy(d_partCord,partCord,  partCordMem, hipMemcpyHostToDevice);
    hipMemcpy(d_ringPara,ringPara,  paraMem,     hipMemcpyHostToDevice);
    
    // each particle is locate at each thread on GPU. 
    dim3 block(128);
    dim3 grid((macroEleNumPerBunch + block.x -1)/ block.x);

    GPU_OneTurnMap<<<grid, block>>>(macroEleNumPerBunch,d_partCord,d_ringPara);

    hipMemcpy(partCord, d_partCord, partCordMem, hipMemcpyDeviceToHost);
    hipFree(d_partCord);
    hipFree(d_ringPara);

}