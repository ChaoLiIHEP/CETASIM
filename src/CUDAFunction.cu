#include "hip/hip_runtime.h"
//*************************************************************************
//Copyright (c) 2020 IHEP                                                  
//Copyright (c) 2021 DESY                                                  
//This program is free software; you can redistribute it and/or modify     
//it under the terms of the GNU General Public License                     
//Author: chao li, li.chao@desy.de                                         
//*************************************************************************
#pragma once                                                             

#include <vector>
#include <complex>
#include <iostream>
#include<fstream>	
#include <numeric>
#include <cmath>
#include "Global.h"
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfft.h>
#include "Global.h"
#include "CUDAFunction.cuh"
#include "hip/hip_math_constants.h"
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>



using namespace std;
using std::vector;
using std::complex;

__global__ void GPU_OneTurnMap(int partNum, double *partCord, double *ringPara )
{
    // partCord {x,px,y,py,z,pz,...}

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // from ringPara input
    double alpha[2]  = {ringPara[0],ringPara[1]};
    double beta[2]   = {ringPara[2],ringPara[3]};
    double nu[2]     = {ringPara[4],ringPara[5]};
    double chrom[2]  = {ringPara[6],ringPara[7]};
    double eta[2]    = {ringPara[8],ringPara[9]};
    double etap[2]   = {ringPara[10],ringPara[11]};
    double aDTX[2]   = {ringPara[12],ringPara[13]};
    double aDTY[2]   = {ringPara[14],ringPara[15]};
    double aDTXY[2]  = {ringPara[16],ringPara[17]};
    double alphac[3] = {ringPara[18],ringPara[19],ringPara[20]};
    double circRing  = ringPara[21];
  

    // // temp during calculation
    double amp[2],nutmp[2],phi[2],gamma[2],sinPhi[2],cosPhi[2];  // used in x and y direction
    double oneTurnMap[6][6];
    memset(oneTurnMap,0,sizeof(oneTurnMap));
    double vectX[6]={0,0,0,0,0,0};
    double vectY[6]={0,0,0,0,0,0};
    int partIndex = idx;


    if(partIndex < partNum)
    {
        #pragma unroll 
        for(int j=0;j<6;j++) vectX[j]  = partCord[partIndex * 6 + j];   // set the vectX (x,px,y,py,z,pz)
                     
        vectX[0] -= eta[0]  * vectX[5];   //x 
        vectX[1] -= etap[0] * vectX[5];   //px
        vectX[2] -= eta[1]  * vectX[5];   //y
        vectX[3] -= etap[1] * vectX[5];   //py

    
        #pragma unroll 
        for(int j=0;j<2;j++)
        {           
            amp[j]  = ( vectX[2*j] * vectX[2*j] + (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) * (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) ) / beta[j];
        }
        
        nutmp[0] = nu[0] + chrom[0] * vectX[5] +  aDTX[0] * amp[0] + aDTX[1] * amp[0] * amp[0]  / 2 + aDTXY[0] * amp[0] * amp[1];
        nutmp[1] = nu[1] + chrom[1] * vectX[5] +  aDTY[0] * amp[1] + aDTY[1] * amp[1] * amp[1]  / 2 + aDTXY[1] * amp[0] * amp[1];
        
        #pragma unroll
        for(int j=0;j<2;j++)
        {
            phi[j]      = 2.0  * M_PI * nutmp[j];
            sinPhi[j]   = sin( phi[j] );
            cosPhi[j]   = cos( phi[j] );
            gamma[j]    = (1 + alpha[j] * alpha[j] ) / beta[j];    
        }
        

        oneTurnMap[0][0] = cosPhi[0] + alpha[0] * sinPhi[0];
        oneTurnMap[0][1] =              beta[0] * sinPhi[0];
        oneTurnMap[1][0] =           - gamma[0] * sinPhi[0];
        oneTurnMap[1][1] = cosPhi[0] - alpha[0] * sinPhi[0];


        oneTurnMap[2][2] = cosPhi[1] + alpha[1] * sinPhi[1];
        oneTurnMap[2][3] =              beta[1] * sinPhi[1];
        oneTurnMap[3][2] =           - gamma[1] * sinPhi[1];
        oneTurnMap[3][3] = cosPhi[1] - alpha[1] * sinPhi[1];

        
        oneTurnMap[0][5] =  eta[0]  -  eta[0] * cosPhi[0] - (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[2][5] =  eta[1]  -  eta[1] * cosPhi[1] - (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];
        oneTurnMap[1][5] = -etap[0] - etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[3][5] = -etap[1] - etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][0] = -etap[0] + etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[4][2] = -etap[1] + etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][1] = eta[0]   - eta[0] * cosPhi[0] + (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[4][3] = eta[1]   - eta[1] * cosPhi[1] + (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];


        oneTurnMap[4][4] = 1;
        oneTurnMap[5][5] = 1;

        #pragma unroll
        for(int j=0;j<6;j++)
        {
            #pragma unroll
            for(int k=0;k<6;k++)
            {
                vectY[j] += oneTurnMap[j][k] * vectX[k];
            }
        }
        vectY[4] -= circRing * (alphac[0] * vectY[5]  + alphac[1] * vectY[5] * vectY[5] + alphac[2] * vectY[5] * vectY[5] * vectY[5] );
        
        #pragma unroll
        for(int j=0;j<6;j++) partCord[partIndex * 6 + j] = vectY[j];
    }
    
}

void GPU_PartiOneTurnTransfer(int totalPartiNum, double *partCord, int paraNum,double *ringPara)
{
    
    int partCordMem = totalPartiNum * 6 * sizeof(double);
    int paraMem     = paraNum *  sizeof(double); 
    double *d_partCord,*d_ringPara;
    
    hipMalloc((void**) &d_partCord, partCordMem);
    hipMalloc((void**) &d_ringPara, paraMem);
    hipMemcpy(d_partCord,partCord,  partCordMem, hipMemcpyHostToDevice);
    hipMemcpy(d_ringPara,ringPara,  paraMem,     hipMemcpyHostToDevice);
    
    // each particle is locate at each thread on GPU. 
    int blockDim;
    if(totalPartiNum>1000) 
    {
        blockDim =128;
    }
    else
    {
        int count = 0;
        int temp=totalPartiNum;
        while(temp/2!=0)
        {
            count++;
            temp /=2;    
        }
        blockDim = pow(2,count);
    }
  
    dim3 block(blockDim);
    dim3 grid((totalPartiNum + block.x -1)/ block.x);

    GPU_OneTurnMap<<<grid, block>>>(totalPartiNum,d_partCord,d_ringPara);

    hipMemcpy(partCord, d_partCord, partCordMem, hipMemcpyDeviceToHost);
    hipFree(d_partCord);
    hipFree(d_ringPara);

}


__global__ void GPU_OneTurnSynRad(int partNum, double *partCord, double *radMatrixBRH, double *radCoeff, double *gaussRand)
{
    // partCord {x,px,y,py,z,pz,...}hiprandStateXORWOW_t  hiprandStateXORWOW_t* states

    int pId = blockIdx.x * blockDim.x + threadIdx.x;

    if(pId < partNum)
    {
        double vectX[6]={0,0,0,0,0,0};
        double vectY[6]={0,0,0,0,0,0};

        #pragma unroll 
        for(int i=0;i<6;i++) vectX[i]  = partCord[pId * 6 + i];   // set the vectX (x,px,y,py,z,pz)
        
        // transfer to norm cord from acc cord
        #pragma unroll
        for(int i=0;i<6;i++)
        {
            #pragma unroll
            for(int j=0;j<6;j++)
            {
                vectY[i] += radMatrixBRH[6*i +j] * vectX[j];
            }
        }
        
        vectY[0] *=  radCoeff[0];
        vectY[1] *=  radCoeff[0];
        vectY[2] *=  radCoeff[1];
        vectY[3] *=  radCoeff[1];
        vectY[5] *=  radCoeff[2];

        
        vectY[0] += radCoeff[3] * gaussRand[pId * 6 + 0]; 
        vectY[1] += radCoeff[3] * gaussRand[pId * 6 + 1]; 
        vectY[2] += radCoeff[4] * gaussRand[pId * 6 + 2]; 
        vectY[3] += radCoeff[4] * gaussRand[pId * 6 + 3]; 
        // vectY[4] += radCoeff[5] * gaussRand[pId * 6 + 4]; 
        vectY[5] += radCoeff[5] * gaussRand[pId * 6 + 5]; 

        // double gaussRand = hiprand_normal_double(state);
        // printf("%15.8f\n",gaussRand[pId * 6]);

        #pragma unroll
        for(int i=0;i<6;i++)
        {
            vectX[i] = 0;
            #pragma unroll
            for(int j=0;j<6;j++)
            {
                vectX[i] += radMatrixBRH[6*i +j + 36] * vectY[j];
            }
        }
         
        // transfer to acc cord from norm cord
        #pragma unroll
        for(int j=0;j<6;j++) partCord[pId * 6 + j] = vectX[j];
    }
    
}


void GPU_PartiSynRad(int totalPartiNum, double *partCord, int paraNum, double *radMatrixBRH, double *radCoeff)
{
    int partCordMem      = totalPartiNum * 6 * sizeof(double);
    int radMatrixBRHMem  = paraNum *  sizeof(double);
    int radCeoffMem      = 6 * sizeof(double);
    
    std::random_device rd{};
    std::mt19937 gen{rd()};
    std::normal_distribution<> dx{0,1};

    double gaussRand[totalPartiNum*6];
    for(int i=0;i<totalPartiNum*6;i++) gaussRand[i    ] = dx(gen);
    for(int i=0;i<totalPartiNum;  i++) gaussRand[6*i+4] = 0;
    
    double *d_partCord, *d_radMatrixBRH,*d_radCoeff,*d_gaussRand;
    hipMalloc((void**) &d_partCord,     partCordMem);
    hipMalloc((void**) &d_gaussRand,    partCordMem);
    hipMalloc((void**) &d_radMatrixBRH, radMatrixBRHMem);
    hipMalloc((void**) &d_radCoeff,     radCeoffMem);
    

    hipMemcpy(d_partCord,     partCord,      partCordMem,         hipMemcpyHostToDevice);
    hipMemcpy(d_gaussRand,    gaussRand,     partCordMem,         hipMemcpyHostToDevice);
    hipMemcpy(d_radMatrixBRH, radMatrixBRH,  radMatrixBRHMem,     hipMemcpyHostToDevice);
    hipMemcpy(d_radCoeff,     radCoeff,      radCeoffMem,         hipMemcpyHostToDevice);  

    int blockDim;
    if(totalPartiNum>1000) 
    {
        blockDim =128;
    }
    else
    {
        int count = 0;
        int temp=totalPartiNum; 
        while(temp/2!=0)
        {
            count++;
            temp /=2;    
        }
        blockDim = pow(2,count);
    }

    dim3 block(blockDim);
    dim3 grid((totalPartiNum + block.x -1)/ block.x);

    GPU_OneTurnSynRad<<<grid, block>>>(totalPartiNum,d_partCord,d_radMatrixBRH,d_radCoeff,d_gaussRand);

    hipMemcpy(partCord, d_partCord, partCordMem, hipMemcpyDeviceToHost);
    // cout<<"host"<<endl;
    // getchar();
    hipFree(d_partCord);
    hipFree(d_radMatrixBRH);
    hipFree(d_radCoeff);
}


__global__ void  GPU_OneTurnMapAndSynRad(int partNum,double *partCord,double *ringPara,double *radMatrixBRH,double *radCoeff,double *gaussRand)
{

    int pId = blockIdx.x * blockDim.x + threadIdx.x;

    if(pId < partNum)
    {
        // one turn map transfer 
        double alpha[2]  = {ringPara[0],ringPara[1]};
        double beta[2]   = {ringPara[2],ringPara[3]};
        double nu[2]     = {ringPara[4],ringPara[5]};
        double chrom[2]  = {ringPara[6],ringPara[7]};
        double eta[2]    = {ringPara[8],ringPara[9]};
        double etap[2]   = {ringPara[10],ringPara[11]};
        double aDTX[2]   = {ringPara[12],ringPara[13]};
        double aDTY[2]   = {ringPara[14],ringPara[15]};
        double aDTXY[2]  = {ringPara[16],ringPara[17]};
        double alphac[3] = {ringPara[18],ringPara[19],ringPara[20]};
        double circRing  = ringPara[21];


        // // temp during calculation
        double amp[2],nutmp[2],phi[2],gamma[2],sinPhi[2],cosPhi[2];  // used in x and y direction
        double oneTurnMap[6][6];
        memset(oneTurnMap,0,sizeof(oneTurnMap));
        double vectX[6]={0,0,0,0,0,0};
        double vectY[6]={0,0,0,0,0,0};
        
        #pragma unroll 
        for(int j=0;j<6;j++) vectX[j]  = partCord[pId * 6 + j];   // set the vectX (x,px,y,py,z,pz)
                     
        vectX[0] -= eta[0]  * vectX[5];   //x 
        vectX[1] -= etap[0] * vectX[5];   //px
        vectX[2] -= eta[1]  * vectX[5];   //y
        vectX[3] -= etap[1] * vectX[5];   //py

    
        #pragma unroll 
        for(int j=0;j<2;j++)
        {           
            amp[j]  = ( vectX[2*j] * vectX[2*j] + (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) * (alpha[j]*vectX[2*j] + beta[j]*vectX[2*j+1]) ) / beta[j];
        }
        
        nutmp[0] = nu[0] + chrom[0] * vectX[5] +  aDTX[0] * amp[0] + aDTX[1] * amp[0] * amp[0]  / 2 + aDTXY[0] * amp[0] * amp[1];
        nutmp[1] = nu[1] + chrom[1] * vectX[5] +  aDTY[0] * amp[1] + aDTY[1] * amp[1] * amp[1]  / 2 + aDTXY[1] * amp[0] * amp[1];
        
        #pragma unroll
        for(int j=0;j<2;j++)
        {
            phi[j]      = 2.0  * M_PI * nutmp[j];
            sinPhi[j]   = sin( phi[j] );
            cosPhi[j]   = cos( phi[j] );
            gamma[j]    = (1 + alpha[j] * alpha[j] ) / beta[j];    
        }
        

        oneTurnMap[0][0] = cosPhi[0] + alpha[0] * sinPhi[0];
        oneTurnMap[0][1] =              beta[0] * sinPhi[0];
        oneTurnMap[1][0] =           - gamma[0] * sinPhi[0];
        oneTurnMap[1][1] = cosPhi[0] - alpha[0] * sinPhi[0];


        oneTurnMap[2][2] = cosPhi[1] + alpha[1] * sinPhi[1];
        oneTurnMap[2][3] =              beta[1] * sinPhi[1];
        oneTurnMap[3][2] =           - gamma[1] * sinPhi[1];
        oneTurnMap[3][3] = cosPhi[1] - alpha[1] * sinPhi[1];

        
        oneTurnMap[0][5] =  eta[0]  -  eta[0] * cosPhi[0] - (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[2][5] =  eta[1]  -  eta[1] * cosPhi[1] - (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];
        oneTurnMap[1][5] = -etap[0] - etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[3][5] = -etap[1] - etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][0] = -etap[0] + etap[0] * cosPhi[0] + ( eta[0] + alpha[0] * alpha[0] * eta[0] + alpha[0] * beta[0] * etap[0] ) * sinPhi[0] / beta[0];
        oneTurnMap[4][2] = -etap[1] + etap[1] * cosPhi[1] + ( eta[1] + alpha[1] * alpha[1] * eta[1] + alpha[1] * beta[1] * etap[1] ) * sinPhi[1] / beta[1];

        oneTurnMap[4][1] = eta[0]   - eta[0] * cosPhi[0] + (alpha[0] * eta[0] + beta[0] * etap[0]) * sinPhi[0];
        oneTurnMap[4][3] = eta[1]   - eta[1] * cosPhi[1] + (alpha[1] * eta[1] + beta[1] * etap[1]) * sinPhi[1];


        oneTurnMap[4][4] = 1;
        oneTurnMap[5][5] = 1;

        #pragma unroll
        for(int j=0;j<6;j++)
        {
            #pragma unroll
            for(int k=0;k<6;k++)
            {
                vectY[j] += oneTurnMap[j][k] * vectX[k];
            }
        }
        vectY[4] -= circRing * (alphac[0] * vectY[5]  + alphac[1] * vectY[5] * vectY[5] + alphac[2] * vectY[5] * vectY[5] * vectY[5] );
        
        #pragma unroll
        for(int j=0;j<6;j++) partCord[pId * 6 + j] = vectY[j];
        //------------------------end of one turn map--------------------------------------------
        
       
       // synchrontron radiation damping--------------------------------------------- 

        #pragma unroll 
        for(int i=0;i<6;i++) vectX[i]  = partCord[pId * 6 + i];   // set the vectX (x,px,y,py,z,pz)
        
        // transfer to norm cord from acc cord
        #pragma unroll
        for(int i=0;i<6;i++)
        {
            vectY[i] = 0.E0;
            #pragma unroll
            for(int j=0;j<6;j++)
            {
                vectY[i] += radMatrixBRH[6*i +j] * vectX[j];
            }
        }
        
        vectY[0] *=  radCoeff[0];
        vectY[1] *=  radCoeff[0];
        vectY[2] *=  radCoeff[1];
        vectY[3] *=  radCoeff[1];
        vectY[5] *=  radCoeff[2];

        
        vectY[0] += radCoeff[3] * gaussRand[pId * 6 + 0]; 
        vectY[1] += radCoeff[3] * gaussRand[pId * 6 + 1]; 
        vectY[2] += radCoeff[4] * gaussRand[pId * 6 + 2]; 
        vectY[3] += radCoeff[4] * gaussRand[pId * 6 + 3]; 
        // vectY[4] += radCoeff[5] * gaussRand[pId * 6 + 4]; 
        vectY[5] += radCoeff[5] * gaussRand[pId * 6 + 5]; 

        // double gaussRand = hiprand_normal_double(state);
        // printf("%15.8f\n",gaussRand[pId * 6]);

        #pragma unroll
        for(int i=0;i<6;i++)
        {
            vectX[i] = 0;
            #pragma unroll
            for(int j=0;j<6;j++)
            {
                vectX[i] += radMatrixBRH[6*i +j + 36] * vectY[j];
            }
        }
         
        // transfer to acc cord from norm cord
        #pragma unroll
        for(int j=0;j<6;j++) partCord[pId * 6 + j] = vectX[j];
        //----------syn end-----------------------------------------------------------------------------------------------
        
    }

}

void GPU_PartiOneTurnTransferAndSynRad(int totalPartiNum, double *partCord, int oneTurnMatrixParaNum, double *ringPara, double *radMatrixBRH, double *radCoeff)
{

    int partCordMem              = totalPartiNum * 6    * sizeof(double);
    int ringParaMem              = oneTurnMatrixParaNum * sizeof(double);
    int radMatrixBRHMem          = 72                   * sizeof(double);
    int radCeoffMem              = 6                    * sizeof(double); 
    
    double *d_partCord,*d_ringPara,*d_radMatrixBRH,*d_radCoeff,*d_gaussRand ;
    
    hipMalloc((void**) &d_partCord,     partCordMem);
    hipMalloc((void**) &d_gaussRand,    partCordMem);
    hipMalloc((void**) &d_ringPara,     ringParaMem);
    hipMalloc((void**) &d_radMatrixBRH, radMatrixBRHMem);
    hipMalloc((void**) &d_radCoeff,     radCeoffMem);


    std::random_device rd{};
    std::mt19937 gen{rd()};
    std::normal_distribution<> dx{0,1};

    double gaussRand[totalPartiNum*6];
    for(int i=0;i<totalPartiNum*6;i++) gaussRand[i    ] = dx(gen);
    for(int i=0;i<totalPartiNum;  i++) gaussRand[6*i+4] = 0;
    

    hipMemcpy(d_partCord,     partCord,      partCordMem,              hipMemcpyHostToDevice);
    hipMemcpy(d_gaussRand,    gaussRand,     partCordMem,              hipMemcpyHostToDevice);
    hipMemcpy(d_ringPara,     ringPara,      ringParaMem,              hipMemcpyHostToDevice);
    hipMemcpy(d_radMatrixBRH, radMatrixBRH,  radMatrixBRHMem,          hipMemcpyHostToDevice);
    hipMemcpy(d_radCoeff,     radCoeff,      radCeoffMem,              hipMemcpyHostToDevice);  
    
    // each particle is locate at each thread on GPU. 
    int blockDim;
    if(totalPartiNum>1000) 
    {
        blockDim =128;
    }
    else
    {
        int count = 0;
        int temp=totalPartiNum;
        while(temp/2!=0)
        {
            count++;
            temp /=2;    
        }
        blockDim = pow(2,count);
    }
    dim3 block(blockDim);
    dim3 grid((totalPartiNum + block.x -1)/ block.x);

    GPU_OneTurnMapAndSynRad<<<grid, block>>>(totalPartiNum,d_partCord,d_ringPara,d_radMatrixBRH,d_radCoeff,d_gaussRand);


    hipMemcpy(partCord, d_partCord, partCordMem, hipMemcpyDeviceToHost);
    hipFree(d_partCord);
    hipFree(d_ringPara);

}